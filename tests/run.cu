#include "run.h"
#include <iostream> 

#include <hip/hip_runtime.h> 
int main() {

	int driver_version;
	hipDriverGetVersion(&driver_version);
	int runtime_version;
	hipRuntimeGetVersion(&runtime_version);
	std::cout << "Runtime version: " << runtime_version  << std::endl;
	std::cout << "Driver version: " << driver_version << std::endl; 
	BC::tests::run();
}
