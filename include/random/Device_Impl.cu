/*
 * Device_Impl.h
 *
 *  Created on: Dec 3, 2018
 *      Author: joseph
 */

#ifdef __HIPCC__
#ifndef BC_RANDOM_DEVICE_DEVICE_IMPL_H_
#define BC_RANDOM_DEVICE_DEVICE_IMPL_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

namespace BC {
namespace random {
namespace device_impl {


__global__
static void bc_curand_init(hiprandState_t* state) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i==0)
    	hiprand_init(0,0,0,state);
}

static constexpr unsigned float_decimal_length = 100000;
template<class T> __global__
static void randomize(hiprandState_t* state, T t, float lower_bound, float upper_bound) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < t.size(); i += blockDim.x * gridDim.x) {
    	hiprandState_t tmpstate = *state;
    	skipahead(i, &tmpstate);
        t[i] = hiprand(&tmpstate) % float_decimal_length;
        t[i] /= float_decimal_length;
        t[i] *= (upper_bound - lower_bound);
        t[i] += lower_bound;
    }

    __syncthreads();
    if (i == 0)
    	skipahead(t.size(), state);
}
}
}
}

#endif
#endif /* DEVICE_IMPL_H_ */
