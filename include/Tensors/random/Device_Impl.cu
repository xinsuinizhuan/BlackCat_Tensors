#include "hip/hip_runtime.h"
/*
 * Device_Impl.h
 *
 *  Created on: Dec 3, 2018
 *      Author: joseph
 */

#ifdef __HIPCC__
#ifndef BC_RANDOM_DEVICE_DEVICE_IMPL_H_
#define BC_RANDOM_DEVICE_DEVICE_IMPL_H_

#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

namespace BC {
namespace random {
namespace device_impl {
template<class T> __global__
static void randomize(T t, float lower_bound, float upper_bound, int seed) {

     hiprandState_t state;
      hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
                  seed, /* the sequence number is only important with multiple cores */
                  1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
                  &state);

    constexpr int floating_point_decimal_length = 10000;

    for (int i = 0; i < t.size(); ++i) {
        t[i] = hiprand(&state) % floating_point_decimal_length;
        t[i] /= floating_point_decimal_length;
        t[i] *= (upper_bound - lower_bound);
        t[i] += lower_bound;
    }
}
}
}
}
#endif
#endif /* DEVICE_IMPL_H_ */
