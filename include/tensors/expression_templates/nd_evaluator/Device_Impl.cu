/*  Project: BlackCat_Tensors
 *  Author: JosephJaspers
 *  Copyright 2018
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/. */

#ifdef __HIPCC__
#ifndef BC_GPU_IMPL
#define BC_GPU_IMPL

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

namespace BC {
namespace tensors {
namespace exprs {
namespace evaluator {
namespace gpu_impl {

template<class T> __global__
static void eval(T t) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < t.size(); i += blockDim.x * gridDim.x) {
        t[i];
    }
}

template<typename T> __global__  static void eval2d(T t) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    for (; n < t.cols(); n += blockDim.x * gridDim.x) {

        int m = blockIdx.y * blockDim.y + threadIdx.y;
        for (; m < t.rows(); m += blockDim.y * gridDim.y) {
            t(n, m);
        }
    }
}
template<typename T> __global__ static void eval3d(T t) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    for (; k < t.dimension(2); k += blockDim.x * gridDim.x) {

    	int n = blockIdx.y * blockDim.y + threadIdx.y;
        for (; n < t.cols(); n += blockDim.y * gridDim.y) {

            int m = blockIdx.z * blockDim.z + threadIdx.z;
            for (; m < t.rows(); m += blockDim.z * gridDim.z) {
                t(k,n,m);
            }
        }
    }
}
//dont know how to do this
template<typename T> __global__ static void eval4d(T t) {
    int l = blockIdx.x * blockDim.x + threadIdx.x;
    for (; l < t.dimension(3); l += blockDim.x * gridDim.x) {

        int k = blockIdx.y * blockDim.y + threadIdx.y;
        for (;k < t.dimension(2); k += blockDim.y * gridDim.y) {

            int n = blockIdx.z * blockDim.z + threadIdx.z;
            for (; n < t.cols(); n += blockDim.z * gridDim.z) {

                for (int m = 0; m < t.rows(); ++m) {
                    t(l, k, n, m);
                }
            }
        }
    }
}
//don't know how to do this
template<typename T> __global__ static void eval5d(T t) {
    int p = blockIdx.z * blockDim.x + threadIdx.x;
    for (; p < t.dimension(4); p += blockDim.x * gridDim.x) {

        int l = blockIdx.y * blockDim.y + threadIdx.y;
        for (; l < t.dimension(3); l += blockDim.y * gridDim.y) {

            int k = blockIdx.x * blockDim.z + threadIdx.z;
            for (; k < t.dimension(2); k += blockDim.z * gridDim.z) {

                for (int n = 0; n < t.dimension(1); ++n) {

                    for (int m = 0; m < t.dimension(0); ++m) {
                        t(p, l, k, n, m);
                    }
                }
            }
        }
    }
}

}
}
}
}
}

#endif
#endif //cudacc
